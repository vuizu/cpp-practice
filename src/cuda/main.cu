
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_runtime_api.h>

__global__ void name_gpu() {
    printf("hello world\n");
}

int main() {

    name_gpu<<<2, 4>>>();
    hipDeviceSynchronize();
}
